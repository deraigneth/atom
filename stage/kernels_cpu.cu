#include "hip/hip_runtime.h"
#include "kernel_cpu.h"

double atomicADD_CPU(double *address, double val) {
  double assumed, old = *address;
  do {
      assumed = old;
      old = __longlong_as_double(atomicCAS((unsigned long long int *)address, __double_as_longlong(assumed), __double_as_longlong(val + assumed)));
  } while (assumed != old);
  return old;
}


void SetAllCurrentsToZero_CPU(GPUCell **cells) {

}

void WriteControlSystem_CPU(Cell **cells) {

}

void writeCurrentComponent_CPU(CellDouble *J, CurrentTensorComponent *t1, CurrentTensorComponent *t2, int pqr2) {

}


void assignSharedWithLocal_CPU(
        CellDouble **c_jx,
        CellDouble **c_jy,
        CellDouble **c_jz,
        CellDouble **c_ex,
        CellDouble **c_ey,
        CellDouble **c_ez,
        CellDouble **c_hx,
        CellDouble **c_hy,
        CellDouble **c_hz,
        CellDouble *fd) {

}

void MoveParticlesInCell_CPU(Cell *c, int index, int blockDimX) {

}

void StepAllCells_CPU(GPUCell **cells) {

}


void emh2_CPU(GPUCell **cells, int i_s, int l_s, int k_s, double *Q, double *H, dim3 dimGrid, dim3 dimBlock) {
  Cell *c0 = cells[0] ;
  for (int i=0 ; i< dimGrid.x; i++){
    for (int j=0 ; j < dimGrid.y ; j++){
      for (int z=0 ; z < dimGrid.z ; z++){
              emh2_Element_CPU(c0, i_s + i, l_s + j, k_s + z, Q, H);
      }
    }
  }
}


void emeElement_CPU(Cell *c, int3 i, double *E, double *H1, double *H2, double *J, double c1, double c2, double tau, int3 d1, int3 d2) {

}

void periodicCurrentElement_CPU(Cell *c, int i, int k, double *E, int dir, int dirE, int N) {


}
void SetFieldsToCells_CPU(GPUCell **cells, double *Ex, double *Ey, double *Ez, double *Hx, double *Hy, double *Hz, dim3 dimGrid, dim3 dimBlockExt){
  Cell *c, *c0 = cells[0] ;
  for (int i=0 ; i< dimGrid.x; i++){
    for (int j=0 ; j < dimGrid.y ; j++){
      for (int z=0 ; z < dimGrid.z ; z++){
              c = cells[c0->getGlobalCellNumber(i,j,z)] ;
      }
    }
  }
  for (int i=0 ; i< dimGrid.x; i++){
    for (int j=0 ; j < dimGrid.y ; j++){
      for (int z=0 ; z < dimGrid.z ; z++){
        for (int g=0 ; g < dimBlockExt.x ; g++){
          for (int h=0 ; h < dimBlockExt.y ; h++){
            for (int k=0 ; k < dimBlockExt.z ; k++){
              uint3 val(g,h,k) ;
              c->readFieldsFromArrays(Ex,Ey,Ez,Hx,Hy,Hz,val) ;
            }
          }
        }
      }
    }
  }
}



void MakeDepartureLists_CPU(GPUCell **cells, int *d_stage){

}

void copyCellDouble_CPU(CellDouble *dst, CellDouble *src, unsigned int n) {

}
void copyFieldsToSharedMemory_CPU(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        CellDouble *c_ex,
        CellDouble *c_ey,
        CellDouble *c_ez,
        CellDouble *c_hx,
        CellDouble *c_hy,
        CellDouble *c_hz,
        Cell *c,
        int index,
        dim3 blockId,
        int blockDimX
){

}
void AccumulateCurrentWithParticlesInCell_CPU(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index, int blockDimX){

}

void CurrentsAllCells_CPU(GPUCell **cells){

}

void emh1_Element_CPU(Cell *c, int3 i, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2) {

}

void periodicElement_CPU(Cell *c, int i, int k, double *E, int dir, int to, int from){

}
 void CurrentPeriodic_CPU(GPUCell **cells, double *E, int dirE, int dir, int i_s, int k_s, int N){

 }

void getCellEnergy_CPU(GPUCell **cells, double *d_ee, double *d_Ex, double *d_Ey, double *d_Ez){
  // à voir plus tard
}

void writeAllCurrents_CPU(GPUCell **cells, int n0, double *jx, double *jy, double *jz, double *rho,dim3 dimGrid, dim3 dimBlock){
  Cell *c, *c0 = cells[0];
  for (int i=0; i<dimGrid.x;i++){
    for (int j=0; j<dimGrid.y;j++){
      for (int k=0; k<dimGrid.z;k++){
        c = cells[c0->getGlobalCellNumber(i,j,k)] ;
      }
    }
  }
  for (int i=0 ; i< dimGrid.x; i++){
    for (int j=0 ; j < dimGrid.y ; j++){
      for (int z=0 ; z < dimGrid.z ; z++){
        for (int g=0 ; g < dimBlockExt.x ; g++){
          for (int h=0 ; h < dimBlockExt.y ; h++){
            for (int p=0 ; p < dimBlockExt.z ; p++){
              int n = c->getFortranCellNumber(c->i + g - 1, c->l + h -1 , c->k + p -1);
              if (n < 0) n = -n;
              double t, t_x, t_y;
              t_x = c->Jx->M[g][h][p];
              int3 i3 = c->getCellTripletNumber(n);

              atomicADD_CPU(&(jx[n]), t_x);
              t_y = c->Jy->M[g][h][p];
              atomicADD_CPU(&(jy[n]), t_y);
              t = c->Jz->M[g][h][p];
              atomicADD_CPU(&(jz[n]), t);
            }
          }
        }
      }
    }
  }

}

void setCellDoubleToZero_CPU(CellDouble *dst, unsigned int n){
  if (n < CellExtent * CellExtent * CellExtent) {
      double *d_dst;

      d_dst = (double *) (dst->M);
      d_dst[n] = 0.0;
  }//a regarder
}

void set_cell_double_arrays_to_zero_CPU(CellDouble *, CellDouble *, CellDouble *, int, int, int){
  //à completer

}

void copyFromSharedMemoryToCell_CPU(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        Cell *c,
        int index
){
  //à completer
}

void emh2_Element_CPU(Cell *c, int i, int l, int k, double *Q, double *H){
  // à compléter
}

void emh1_CPU(GPUCell **cells, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2){
  //à compléter
}

void periodicCurrentElement_CPU(Cell *c, int i, int k, double *E, int dir, int dirE, int N){
  //à compléter

}

void periodic_CPU(GPUCell **cells, int i_s, int k_s, double *E, int dir, int to, int from) {

}

void eme_CPU(GPUCell **cells, int3 s, double *E, double *H1, double *H2, double *J, double c1, double c2, double tau, int3 d1, int3 d2, dim3 dimGrid, dim3 dimBlock){
  Cell *c0 = cells[0];
  // for nx
  for (int i=0; i<dimGrid.x;i++){
    for (int j = 0 ; j < dimBlock.x ; j++){
      s.x += j ;
    }
    s.x += i ;
  }
  // ou alors s.x = i * dimGrid.x + j ;

  // for ny
  for (int i=0; i<dimGrid.y;i++){
    for (int j = 0 ; j < dimBlock.y ; j++){
      s.y += j ;
    }
    s.y += i ;
  }

  // for nz
  for (int i=0; i<dimGrid.z;i++){
    for (int j = 0 ; j < dimBlock.z ; j++){
      s.z += j ;
    }
    s.z += i ;
  }

  // à changer car fonction GPU
  // emeElement_GPU(c0, s, E, H1, H2, J, c1, c2, tau, d1, d2);


}
