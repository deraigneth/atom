#include "hip/hip_runtime.h"
#include "kernel_cpu.h"

double atomicADD_CPU(double *address, double val) {
  double assumed, old = *address;
  do {
      assumed = old;
      old = __longlong_as_double(atomicCAS((unsigned long long int *)address, __double_as_longlong(assumed), __double_as_longlong(val + assumed)));
  } while (assumed != old);
  return old;
}

void getCellEnergy_CPU(GPUCell **cells, double *d_ee, double *d_Ex, double *d_Ey, double *d_Ez){
  // à voir plus tard
}

void writeAllCurrents_CPU(GPUCell **cells, int n0, double *jx, double *jy, double *jz, double *rho){
  //à voir plus tard
}

void setCellDoubleToZero_CPU(CellDouble *dst, unsigned int n){
  if (n < CellExtent * CellExtent * CellExtent) {
      double *d_dst;

      d_dst = (double *) (dst->M);
      d_dst[n] = 0.0;
  }//a regarder
}

void set_cell_double_arrays_to_zero_CPU(CellDouble *, CellDouble *, CellDouble *, int, int, int){
  //à completer

}

void copyFromSharedMemoryToCell_CPU(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        Cell *c,
        int index
){
  //à completer
}

void emh2_Element_CPU(Cell *c, int i, int l, int k, double *Q, double *H){
  // à compléter
}

void emh1_CPU(GPUCell **cells, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2){
  //à compléter
}

void periodicCurrentElement_CPU(Cell *c, int i, int k, double *E, int dir, int dirE, int N){
  //à compléter
}
