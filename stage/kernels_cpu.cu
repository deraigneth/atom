#include "hip/hip_runtime.h"
#include "kernel_cpu.h"

double atomicADD_CPU(double *address, double val) {
  double assumed, old = *address;
  do {
      assumed = old;
      old = __longlong_as_double(atomicCAS((unsigned long long int *)address, __double_as_longlong(assumed), __double_as_longlong(val + assumed)));
  } while (assumed != old);
  return old;
}

void SetFieldsToCells_CPU(GPUCell **cells, double *Ex, double *Ey, double *Ez, double *Hx, double *Hy, double *Hz){

}

void MakeDepartureLists_CPU(GPUCell **cells, int *d_stage){

}

void copyCellDouble_CPU(CellDouble *dst, CellDouble *src, unsigned int n) {

}
void copyFieldsToSharedMemory_CPU(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        CellDouble *c_ex,
        CellDouble *c_ey,
        CellDouble *c_ez,
        CellDouble *c_hx,
        CellDouble *c_hy,
        CellDouble *c_hz,
        Cell *c,
        int index,
        dim3 blockId,
        int blockDimX
){

}
void AccumulateCurrentWithParticlesInCell_CPU(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index, int blockDimX){

}

void CurrentsAllCells_CPU(GPUCell **cells){

}

void emh1_Element_CPU(Cell *c, int3 i, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2) {

}

void periodicElement_CPU(Cell *c, int i, int k, double *E, int dir, int to, int from){

}
 void CurrentPeriodic_CPU(GPUCell **cells, double *E, int dirE, int dir, int i_s, int k_s, int N){

 }
