#ifdef __HIPCC__
int MemoryCopy(void* dst,void *src,size_t size,int dir) {
    hipMemcpyKind cuda_dir;

    if(dir == HOST_TO_DEVICE) cuda_dir = hipMemcpyHostToDevice;
    if(dir == HOST_TO_HOST) cuda_dir = hipMemcpyHostToHost;
    if(dir == DEVICE_TO_HOST) cuda_dir = hipMemcpyDeviceToHost;
    if(dir == DEVICE_TO_DEVICE) cuda_dir = hipMemcpyDeviceToDevice;

    int status = (int)hipMemcpy(dst,src,size,cuda_dir);

    return status != 0 ? -1 : 0;
}
#else
int MemoryCopy(void *dst, void *src, size_t size, int dir) {
    return 0;
}
#endif

#ifdef __HIPCC__
int MemoryAllocate(void** dst,size_t size) {
   int err = hipMalloc(dst, size);

   return err != 0 ? -1 : 0;
}
#else
int MemoryAllocate(void **dst, size_t size) {
    return 0;
}
#endif


#ifdef __HIPCC__
int getLastError() {
    return (int)hipGetLastError();
}
#else
int getLastError() {
    return 0;
}
#endif

#ifdef __HIPCC__
const char *getErrorString(int err) {
   return hipGetErrorString((hipError_t)err);
}
#else
const char *getErrorString(int err) {
    return "";
}
#endif

#ifdef __HIPCC__
int memory_monitor(std::string legend) {
#ifdef DEBUG
    size_t m_free, m_total;
    struct sysinfo info;

    int err = hipMemGetInfo(&m_free, &m_total);
    CHECK_ERROR("hipMemGetInfo", err);

    sysinfo(&info);
    printf("%40s | GPU memory: total %8d MB | free %8d MB | CPU memory: total %8u MB | free %8u MB\n",
           legend.c_str(),
           (int) (m_total / 1024 / 1024),
           (int) (m_free / 1024 / 1024),
           (int) (info.totalram / 1024 / 1024),
           (int) (info.freeram / 1024 / 1024));
#endif

    return 0;
}
#else
int memory_monitor(std::string legend) {
    return 0;
}
#endif

///
