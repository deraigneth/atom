#include "hip/hip_runtime.h"
#ifdef __HIPCC__
double cuda_atomicAdd(double *address, double val) {
  atomicAdd_cuda(double *address, double val) ;
}
#else
double cuda_atomicAdd(double *address, double val) {
  atomicADD_CPU(double *address, double val) ;
}
#endif


#ifdef __HIPCC__
void GPU_getCellEnergy(GPUCell **cells, double *d_ee, double *d_Ex, double *d_Ey, double *d_Ez){
  getCellEnergy_GPU(GPUCell **cells, double *d_ee, double *d_Ex, double *d_Ey, double *d_Ez);
}
#else
void GPU_getCellEnergy(GPUCell **cells, double *d_ee, double *d_Ex, double *d_Ey, double *d_Ez){
  getCellEnergy_CPU(GPUCell **cells, double *d_ee, double *d_Ex, double *d_Ey, double *d_Ez);
}
#endif


#ifdef __HIPCC__
void GPU_WriteAllCurrents(GPUCell **cells, int n0, double *jx, double *jy, double *jz, double *rho){
  writeAllCurrents_GPU(GPUCell **cells, int n0, double *jx, double *jy, double *jz, double *rho);
}
#else
void GPU_WriteAllCurrents(GPUCell **cells, int n0, double *jx, double *jy, double *jz, double *rho){
  writeAllCurrents_CPU(GPUCell **cells, int n0, double *jx, double *jy, double *jz, double *rho);
}
#endif


#ifdef __HIPCC__
void GPU_ArrangeFlights(GPUCell **cells, int *d_stage){
  arrangeFlights_GPU(GPUCell **cells, int *d_stage);
}
#else
void GPU_ArrangeFlights(GPUCell **cells, int *d_stage){
  arrangeFlights_CPU(GPUCell **cells, int *d_stage);
}
#endif


#ifdef __HIPCC__
void setCellDoubleToZero(CellDouble *dst, unsigned int n){
  setCellDoubleToZero_GPU(CellDouble *dst, unsigned int n);
}
#else
void setCellDoubleToZero(CellDouble *dst, unsigned int n){
  setCellDoubleToZero_CPU(CellDouble *dst, unsigned int n);
}

#ifdef __HIPCC__
void set_cell_double_arrays_to_zero(CellDouble *, CellDouble *, CellDouble *, int, int, int){
  set_cell_double_arrays_to_zero_GPU(CellDouble *, CellDouble *, CellDouble *, int, int, int);
}
#else
void set_cell_double_arrays_to_zero(CellDouble *, CellDouble *, CellDouble *, int, int, int){
  set_cell_double_arrays_to_zero_CPU(CellDouble *, CellDouble *, CellDouble *, int, int, int);
}
#endif


#ifdef __HIPCC__
void copyFromSharedMemoryToCell(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        Cell *c,
        int index
){
  copyFromSharedMemoryToCell_GPU(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index);
}
#else
void copyFromSharedMemoryToCell(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        Cell *c,
        int index
){
  copyFromSharedMemoryToCell_CPU(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index);
}
#endif

#ifdef __HIPCC__
void emh2_Element(Cell *c, int i, int l, int k, double *Q, double *H){
  emh2_Element_GPU(Cell *c, int i, int l, int k, double *Q, double *H);
}
#else
void emh2_Element(Cell *c, int i, int l, int k, double *Q, double *H){
  emh2_Element_CPU(Cell *c, int i, int l, int k, double *Q, double *H);
}
#endif


#ifdef __HIPCC__
void GPU_emh1(GPUCell **cells, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2){
  emh1_GPU(GPUCell **cells, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2);
}
#else
void GPU_emh1(GPUCell **cells, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2){
  emh1_CPU(GPUCell **cells, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2);
}
#endif

#ifdef __HIPCC__
void periodicCurrentElement(Cell *c, int i, int k, double *E, int dir, int dirE, int N){
  periodicCurrentElement_GPU(Cell *c, int i, int k, double *E, int dir, int dirE, int N);
}
#else
void periodicCurrentElement(Cell *c, int i, int k, double *E, int dir, int dirE, int N){
  periodicCurrentElement_CPU(Cell *c, int i, int k, double *E, int dir, int dirE, int N);
}
#endif
