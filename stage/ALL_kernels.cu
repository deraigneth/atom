#include "hip/hip_runtime.h"
#ifdef __HIPCC__
double cuda_atomicAdd(double *address, double val) {
  atomicAdd_GPU(double *address, double val) ;
}
#else
double cuda_atomicAdd(double *address, double val) {
  atomicADD_CPU(double *address, double val) ;
}
#endif

#ifdef __HIPCC__
void GPU_SetFieldsToCells(GPUCell **cells, double *Ex, double *Ey, double *Ez, double *Hx, double *Hy, double *Hz){
  SetFieldsToCells_GPU(GPUCell **cells, double *Ex, double *Ey, double *Ez, double *Hx, double *Hy, double *Hz) ;
}
#else
void GPU_SetFieldsToCells(GPUCell **cells, double *Ex, double *Ey, double *Ez, double *Hx, double *Hy, double *Hz){
  SetFieldsToCells_CPU(GPUCell **cells, double *Ex, double *Ey, double *Ez, double *Hx, double *Hy, double *Hz) ;
}
#endif

#ifdef __HIPCC__
void GPU_MakeDepartureLists(GPUCell **cells, int *d_stage){
  MakeDepartureLists_GPU(GPUCell **cells, int *d_stage) ;
}
#else
void GPU_MakeDepartureLists(GPUCell **cells, int *d_stage){
  MakeDepartureLists_CPU(GPUCell **cells, int *d_stage);
}
#endif

#ifdef __HIPCC__
void copyCellDouble(CellDouble *dst, CellDouble *src, unsigned int n){
  copyCellDouble_GPU(CellDouble *dst, CellDouble *src, unsigned int n);
}
#else
void copyCellDouble(CellDouble *dst, CellDouble *src, unsigned int n){
  copyCellDouble_CPU(CellDouble *dst, CellDouble *src, unsigned int n) ;
}
#endif

#ifdef __HIPCC__
void copyFieldsToSharedMemory(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        CellDouble *c_ex,
        CellDouble *c_ey,
        CellDouble *c_ez,
        CellDouble *c_hx,
        CellDouble *c_hy,
        CellDouble *c_hz,
        Cell *c,
        int index,
        dim3 blockId,
        int blockDimX
){
  copyFieldsToSharedMemory_GPU(
          CellDouble *c_jx,
          CellDouble *c_jy,
          CellDouble *c_jz,
          CellDouble *c_ex,
          CellDouble *c_ey,
          CellDouble *c_ez,
          CellDouble *c_hx,
          CellDouble *c_hy,
          CellDouble *c_hz,
          Cell *c,
          int index,
          dim3 blockId,
          int blockDimX
  );
}
#else
void copyFieldsToSharedMemory(
        CellDouble *c_jx,
        CellDouble *c_jy,
        CellDouble *c_jz,
        CellDouble *c_ex,
        CellDouble *c_ey,
        CellDouble *c_ez,
        CellDouble *c_hx,
        CellDouble *c_hy,
        CellDouble *c_hz,
        Cell *c,
        int index,
        dim3 blockId,
        int blockDimX
){
  copyFieldsToSharedMemory_CPU(
          CellDouble *c_jx,
          CellDouble *c_jy,
          CellDouble *c_jz,
          CellDouble *c_ex,
          CellDouble *c_ey,
          CellDouble *c_ez,
          CellDouble *c_hx,
          CellDouble *c_hy,
          CellDouble *c_hz,
          Cell *c,
          int index,
          dim3 blockId,
          int blockDimX
  );
}
#endif

#ifdef __HIPCC__
void AccumulateCurrentWithParticlesInCell(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index, int blockDimX){
  AccumulateCurrentWithParticlesInCell_GPU(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index, int blockDimX);
}
#else
void AccumulateCurrentWithParticlesInCell(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index, int blockDimX){
  AccumulateCurrentWithParticlesInCell_CPU(CellDouble *c_jx, CellDouble *c_jy, CellDouble *c_jz, Cell *c, int index, int blockDimX);
}
#endif

#ifdef __HIPCC__
void GPU_CurrentsAllCells(GPUCell **cells){
  void CurrentsAllCells_GPU(GPUCell **cells);
}
#else
void GPU_CurrentsAllCells(GPUCell **cells){
  void CurrentsAllCells_CPU(GPUCell **cells);
}
#endif

#ifdef __HIPCC__
void emh1_Element(Cell *, int3, double *, double *, double *, double *, double, double, int3, int3){
  emh1_Element_GPU(Cell *c, int3 i, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2) ;
}
#else
void emh1_Element(Cell *, int3, double *, double *, double *, double *, double, double, int3, int3){
  emh1_Element_CPU(Cell *c, int3 i, double *Q, double *H, double *E1, double *E2, double c1, double c2, int3 d1, int3 d2) ;
}
#endif

#ifdef __HIPCC__
void periodicElement(Cell *c, int i, int k, double *E, int dir, int to, int from){
  periodicElement_GPU(Cell *c, int i, int k, double *E, int dir, int to, int from) ;
}
#else
void periodicElement(Cell *c, int i, int k, double *E, int dir, int to, int from){
  periodicElement_CPU(Cell *c, int i, int k, double *E, int dir, int to, int from);
}

#ifdef __HIPCC__
void GPU_CurrentPeriodic(GPUCell **cells, double *E, int dirE, int dir, int i_s, int k_s, int N){
  CurrentPeriodic_GPU(GPUCell **cells, double *E, int dirE, int dir, int i_s, int k_s, int N);
}
#else
void GPU_CurrentPeriodic(GPUCell **cells, double *E, int dirE, int dir, int i_s, int k_s, int N){
  CurrentPeriodic_CPU(GPUCell **cells, double *E, int dirE, int dir, int i_s, int k_s, int N);
}
