#ifdef __HIPCC__
double cuda_atomicAdd(double *address, double val) {
  atomicAdd_cuda(double *address, double val) ;
}
#else
double cuda_atomicAdd(double *address, double val) {
  atomicADD(double *address, double val) ;
}
#endif
